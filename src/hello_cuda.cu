
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel() {
    printf("Hello, World!\n");
}

int main() {
    kernel<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}